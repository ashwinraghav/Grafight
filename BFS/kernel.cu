#include "hip/hip_runtime.h"
/*
   Written By Ashwin Raghav
   Twitter @ashwinraghav
   blog.ashwinraghav.com
   github.com/ashwinraghav
   If you want to copy the code, by all means DO	
 */

#ifndef _KERNEL_H_
#define _KERNEL_H_

__global__ void Kernel( Node* g_graph_nodes, int* g_graph_edges, bool* g_graph_mask, bool* g_graph_visited, int* g_cost, bool *g_over, int no_of_nodes) 
{
	int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
	if( tid<no_of_nodes && g_graph_mask[tid])
	{
		g_graph_mask[tid]=false;
		g_graph_visited[tid]=true;
		for(int i=g_graph_nodes[tid].starting; i<(g_graph_nodes[tid].no_of_edges + g_graph_nodes[tid].starting); i++)
		{
			int id = g_graph_edges[i];
			if(!g_graph_visited[id])
			{
				g_cost[id]=g_cost[tid]+1;
				g_graph_mask[id]=true;
				//Change the loop stop value such that loop continues
				*g_over=true;
			}
		}
	}
}

#endif 
