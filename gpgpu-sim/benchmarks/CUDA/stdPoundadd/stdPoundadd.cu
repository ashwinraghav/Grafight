/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation and 
 * any modifications thereto.  Any use, reproduction, disclosure, or distribution 
 * of this software and related documentation without an express license 
 * agreement from NVIDIA Corporation is strictly prohibited.
 * 
 */

/* A simple program demonstrating trivial use of global memory atomic 
   device functions (atomic*() functions).
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include "cutil_inline.h"

// includes, kernels
#include "stdPoundadd_kernel.cu"

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);

extern "C"
int computeGold( int* gpuData, const int len);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
    runTest( argc, argv);

    cutilExit(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest( int argc, char **argv)
{
    hipDeviceProp_t deviceProp;
    deviceProp.major = 0;
    deviceProp.minor = 0;
    int dev;

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
        cutilDeviceInit(argc, argv);
    else
        hipSetDevice( dev = cutGetMaxGflopsDeviceId() );

    cutilSafeCall( hipChooseDevice(&dev, &deviceProp) );
    cutilSafeCall( hipGetDeviceProperties(&deviceProp, dev) );

    if(deviceProp.major > 1 || deviceProp.minor > 0)
    {
        printf("Using Device %d: \"%s\"\n", dev, deviceProp.name);
    }
    else
    {
        printf("There is no device supporting CUDA compute capability 1.1.\n");
        printf("TEST PASSED");
        hipDeviceReset();
        cutilExit(argc, argv);
    }

    unsigned int timer = 0;
    cutilCheckError( cutCreateTimer( &timer));
    cutilCheckError( cutStartTimer( timer));

    int numThreads = 256;
    unsigned int numBlocks = 64;
    unsigned int numData = 11;
    unsigned int memSize = sizeof(int) * numData;

    if (cutCheckCmdLineFlag(argc, (const char **) argv, "n")) {
      cutGetCmdLineArgumenti( argc, (const char**) argv, "n", &numThreads); }

    //allocate mem for the result on host side
    int *h_odata = (int *)malloc(memSize);

    //initalize the memory
    for(unsigned int i = 0; i < numData; i++)
        h_odata[i] = 0;

    //To make the AND and XOR tests generate something other than 0...
    //h_odata[8] = h_odata[10] = 0xff; 

    // allocate device memory for result
    int *d_odata;
    cutilSafeCall( hipMalloc( (void**) &d_odata, memSize));
    // copy host memory to device to initialize to zers
    cutilSafeCall( hipMemcpy( d_odata, h_odata, memSize, hipMemcpyHostToDevice) );

// Create events to time kernel execution
    float exec_time;
    hipEvent_t start, stop;
    int eventflags = hipEventBlockingSync;

    //    cutilSafeCall( hipEventCreateWithFlags(&start, eventflags) );
    //    cutilSafeCall( hipEventCreateWithFlags(&stop, eventflags) );

 
    // execute the kernel
    //    hipEventRecord(start, 0);
    addKernel<<<numBlocks, numThreads>>>(d_odata);
    //    hipEventRecord(stop, 0);
    //    hipEventSynchronize(stop);
    //    cutilSafeCall(hipEventElapsedTime(&exec_time, start, stop));
    cutilCheckMsg("Kernel execution failed");
    //Copy result from device to host
    //    printf( "Standard Add time: %f (ms)\n", exec_time);
    cutilSafeCall( hipMemcpy( h_odata, d_odata, memSize, hipMemcpyDeviceToHost) );


    //compute reference solution
    if(computeGold(h_odata, numThreads * numBlocks))
        printf("Test PASSED\n");
    else
        printf("Test FAILED\n");

    // cleanup memory
    free(h_odata);
    cutilSafeCall(hipFree(d_odata));

    hipDeviceReset();
}
